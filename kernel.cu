#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "MM Params.h"

struct Int1_t
{
	unsigned int val:1;
};


__global__ void mm(Matrix_t a, Matrix_t b, Matrix_t out)
{
	int out_x = blockIdx.x*block_size + threadIdx.x;
	int out_y = blockIdx.y*block_size + threadIdx.y;

	//Some threads don't produce any value, but they are needed to copy whole blocks to shared memory
	bool has_output = out_x < out.columns && out_y < out.rows;


	//Declare shared blocks
	__shared__ FloatT shared_memory_0[block_size*block_size];
	__shared__ FloatT shared_memory_1[block_size*block_size];
	__shared__ FloatT shared_memory_2[block_size*block_size];
	__shared__ FloatT shared_memory_3[block_size*block_size];

	//Wrap them intro Matrix class
	Matrix_t shared_a[] = { Matrix_t(shared_memory_0, block_size, block_size), Matrix_t(shared_memory_1, block_size, block_size) };
	Matrix_t shared_b[] = { Matrix_t(shared_memory_2, block_size, block_size), Matrix_t(shared_memory_3, block_size, block_size) };

	Int1_t fetch_idx{ 0 };

	//Fetch first block
	{
		//Copy element's value to shared memory or set it to 0 if it doens't exist
		int a_x = 0 * block_size + threadIdx.x;
		int a_y = blockIdx.y*block_size + threadIdx.y;
		int b_x = blockIdx.x*block_size + threadIdx.x;
		int b_y = 0 * block_size + threadIdx.y;

		if (a_x < a.columns && a_y < a.rows)
			shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = a.at(a_y, a_x);
		else
			shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;

		if (b_x < b.columns && b_y < b.rows)
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x) = b.at(b_y, b_x);
		else
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;

		fetch_idx.val++;
	}

	__syncthreads();

	
	FloatT acc = 0;
	for (int block_num = 0, blocks_to_process = std::ceil(static_cast<double>(a.columns) / block_size); block_num < blocks_to_process; block_num++) {//TODO CEIL
		//Fetch next block
		if (block_num+1 != blocks_to_process) {
			int next_block_num = block_num + 1;
			int a_x = next_block_num * block_size + threadIdx.x;
			int a_y = blockIdx.y*block_size + threadIdx.y;
			int b_x = blockIdx.x*block_size + threadIdx.x;
			int b_y = next_block_num * block_size + threadIdx.y;

			//Copy element's value to shared memory or set it to 0 if it doens't exist
			if (a_x < a.columns && a_y < a.rows)
				shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = a.at(a_y, a_x);
			else
				shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;

			if (b_x < b.columns && b_y < b.rows)
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x) = b.at(b_y, b_x);
			else
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;
		}


		//Now we will make use of previously fetched block (and later fetch again to it's index)
		fetch_idx.val++;

		//Accumulate result
		for (int i = 0; i < block_size; i++) 
			acc += shared_a[fetch_idx.val].at(threadIdx.y, i)*shared_b[fetch_idx.val].at(i, threadIdx.x);
		
		__syncthreads();
	}

	if (has_output)
		out.at(out_y, out_x) = acc;
}

__global__ void mm2(Matrix_t a, Matrix_t b, Matrix_t out)
{
	int out_x_0 = blockIdx.x*block_size*2 + threadIdx.x*2;
	int out_x_1 = out_x_0 + 1;
	int out_y = blockIdx.y*block_size + threadIdx.y;


	//Some threads don't produce any value, but they are needed to copy whole blocks to shared memory
	bool has_output_0 = out_x_0 < out.columns && out_y < out.rows;
	bool has_output_1 = out_x_1 < out.columns && out_y < out.rows;


	//Declare shared blocks (B blocks are 2x wider)
	__shared__ FloatT shared_memory_0[block_size*block_size];
	__shared__ FloatT shared_memory_1[block_size*block_size];
	__shared__ FloatT shared_memory_2[block_size*block_size*2];
	__shared__ FloatT shared_memory_3[block_size*block_size*2];

	//Wrap them intro Matrix class
	Matrix_t shared_a[] = { Matrix_t(shared_memory_0, block_size, block_size), Matrix_t(shared_memory_1, block_size, block_size) };
	Matrix_t shared_b[] = { Matrix_t(shared_memory_2, block_size, block_size*2), Matrix_t(shared_memory_3, block_size, block_size*2) };

	Int1_t fetch_idx{ 0 };

	//Fetch first block
	{
		//Copy element's value to shared memory or set it to 0 if it doens't exist
		int a_x = 0 * block_size + threadIdx.x;
		int a_y = blockIdx.y*block_size + threadIdx.y;
		int b_x_0 = blockIdx.x*block_size * 2 + threadIdx.x * 2;
		int b_x_1 = b_x_0 + 1;
		int b_y = 0 * block_size + threadIdx.y;

		if (a_x < a.columns && a_y < a.rows)
			shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = a.at(a_y, a_x);
		else
			shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;

		if (b_x_0 < b.columns && b_y < b.rows)
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x*2) = b.at(b_y, b_x_0);
		else
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x*2) = 0;

		if (b_x_1 < b.columns && b_y < b.rows)
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x*2+1) = b.at(b_y, b_x_1);
		else
			shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x*2+1) = 0;
	}

	fetch_idx.val++;

	__syncthreads();


	FloatT acc_0 = 0;
	FloatT acc_1 = 0;
	for (int block_num = 0, blocks_to_process = std::ceil(static_cast<double>(a.columns) / block_size); block_num < blocks_to_process; block_num++) {
		//Fetch next block																																			 
		if (block_num + 1 != blocks_to_process) {
			int next_block_num = block_num + 1;
			int a_x = next_block_num * block_size + threadIdx.x;
			int a_y = blockIdx.y*block_size + threadIdx.y;
			int b_x_0 = blockIdx.x*block_size * 2 + threadIdx.x * 2;
			int b_x_1 = b_x_0 + 1;
			int b_y = next_block_num * block_size + threadIdx.y;

			//Copy element's value to shared memory or set it to 0 if it doens't exist
			if (a_x < a.columns && a_y < a.rows)
				shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = a.at(a_y, a_x);
			else
				shared_a[fetch_idx.val].at(threadIdx.y, threadIdx.x) = 0;

			if (b_x_0 < b.columns && b_y < b.rows)
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x * 2) = b.at(b_y, b_x_0);
			else
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x * 2) = 0;

			if (b_x_1 < b.columns && b_y < b.rows)
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x * 2 + 1) = b.at(b_y, b_x_1);
			else
				shared_b[fetch_idx.val].at(threadIdx.y, threadIdx.x * 2 + 1) = 0;
		}

		//Now we will make use of previously fetched block (and later fetch again to it's index)
		fetch_idx.val++;

		//Accumulate result
		for (int i = 0; i < block_size; i++)
			acc_0 += shared_a[fetch_idx.val].at(threadIdx.y, i)*shared_b[fetch_idx.val].at(i, threadIdx.x*2);

		for (int i = 0; i < block_size; i++)
			acc_1 += shared_a[fetch_idx.val].at(threadIdx.y, i)*shared_b[fetch_idx.val].at(i, threadIdx.x * 2+1);

		__syncthreads();
	}

	if (has_output_0)
		out.at(out_y, out_x_0) = acc_0;
	if (has_output_1)
		out.at(out_y, out_x_1) = acc_1;
}

Matrix_t cuda_matmul(Matrix_t & h_a, Matrix_t & h_b, bool use_mm2) {
	if (h_a.columns != h_b.rows)
		throw std::runtime_error("Sizes don't match");

	hipError_t cudaStatus;
	FloatT * d_a_memory;
	FloatT * d_b_memory;
	FloatT * d_out_memory;
	Matrix_t h_out(h_a.rows, h_b.columns);

	try {
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipSetDevice failed!Do you have a CUDA - capable GPU installed ? ");

		cudaStatus = hipMalloc((void**)&d_a_memory, h_a.num_elems() * sizeof(FloatT));
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMalloc A failed");

		cudaStatus = hipMalloc((void**)&d_b_memory, h_b.num_elems() * sizeof(FloatT));
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMalloc B failed");

		cudaStatus = hipMalloc((void**)&d_out_memory, h_out.num_elems() * sizeof(FloatT));
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMalloc Out failed");

		cudaStatus = hipMemcpy(d_a_memory, h_a.begin(), h_a.num_elems() * sizeof(FloatT), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMemcpy A failed");

		cudaStatus = hipMemcpy(d_b_memory, h_b.begin(), h_b.num_elems() * sizeof(FloatT), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMemcpy B failed");

		Matrix_t d_a(d_a_memory, h_a.rows, h_a.columns);
		Matrix_t d_b(d_b_memory, h_b.rows, h_b.columns);
		Matrix_t d_out(d_out_memory, h_out.rows, h_out.columns);
		dim3 block(block_size, block_size);
		
		if (!use_mm2) {
			dim3 grid(std::ceil(static_cast<double>(d_out.columns) / block_size),
					  std::ceil(static_cast<double>(d_out.rows) / block_size));
			mm << <grid, block >> > (d_a.shallow_copy(), d_b.shallow_copy(), d_out.shallow_copy());
		}
		else {
			dim3 grid(std::ceil(static_cast<double>(d_out.columns) / block_size / 2),
					  std::ceil(static_cast<double>(d_out.rows) / block_size));
			mm2 << <grid, block >> > (d_a.shallow_copy(), d_b.shallow_copy(), d_out.shallow_copy());
		}


		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("MM kernel launch failed");

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipDeviceSynchronize returned error code %d after launching addKernel!\n");

		cudaStatus = hipMemcpy(h_out.begin(), d_out.begin(), d_out.num_elems() * sizeof(FloatT), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("hipMemcpy to host failed");

		hipFree(d_a_memory);
		hipFree(d_b_memory);
		hipFree(d_out_memory);

		return h_out;
	}
	catch (std::exception & e) {
		std::cerr << e.what() << std::endl;
		std::cerr << "Cuda status: " << cudaStatus << std::endl;
		hipFree(d_a_memory);
		hipFree(d_b_memory);
		hipFree(d_out_memory);
		throw std::runtime_error("Cuda mm failed");
	}


}







//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//    {
//		int matrix_size = 4;
//		Matrix_t h_a(matrix_size, matrix_size); for (FloatT & e : h_a) e = 1;
//		Matrix_t h_b(matrix_size, matrix_size); for (FloatT & e : h_b) e = 1;
//		Matrix_t h_out(matrix_size, matrix_size);
//
//		FloatT* d_a;
//		cudaStatus = hipMalloc((void**)&d_a, matrix_size * matrix_size * sizeof(FloatT));
//		hipMemcpy(d_a, h_a.begin(), h_a.num_elems() * sizeof(FloatT), hipMemcpyHostToDevice);
//		FloatT* d_b;
//		cudaStatus = hipMalloc((void**)&d_b, matrix_size * matrix_size * sizeof(FloatT));
//		hipMemcpy(d_b, h_b.begin(), h_a.num_elems() * sizeof(FloatT), hipMemcpyHostToDevice);
//		FloatT* d_out;
//		cudaStatus = hipMalloc((void**)&d_out, matrix_size * matrix_size * sizeof(FloatT));
//
//
//		Matrix_t a(d_a, matrix_size, matrix_size);
//		Matrix_t b(d_b, matrix_size, matrix_size);
//		Matrix_t out(d_out, matrix_size, matrix_size);
//		// Launch a kernel on the GPU with one thread for each element.
//		mm <<<dim3(1,1), dim3(block_size, block_size)>>>(a.shallow_copy(),b.shallow_copy(),out.shallow_copy());
//		hipDeviceSynchronize();
//		hipMemcpy(h_out.begin(), out.begin(), h_out.num_elems() * sizeof(FloatT), hipMemcpyDeviceToHost);
//		h_out.print();
//    }
//
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
